
#include <hip/hip_runtime.h>
#include "time.h"

constexpr int segment_size = 1024;
constexpr int threads = 512;

void __global__ alloc(int **pointers) {
  auto index = blockIdx.x * blockDim.x + threadIdx.x;
  pointers[index] = (int *)malloc(segment_size);
}

void __global__ fill(int **pointers) {
  auto index = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = 0; i < segment_size / sizeof(int); i++) {
    pointers[index][i] = i;
  }
}

void __global__ free(int **pointers) {
  auto index = blockIdx.x * blockDim.x + threadIdx.x;
  free(pointers[index]);
}

int main() {
  int **pointers;
  hipMalloc(&pointers, threads * sizeof(int *));

  int bd = 32;
  for (int i = 0; i < 10; i++) {
    alloc<<<threads / bd, bd>>>(pointers);
    fill<<<threads / bd, bd>>>(pointers);
    free<<<threads / bd, bd>>>(pointers);
  }
  hipDeviceSynchronize();
}